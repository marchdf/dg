
#include <hip/hip_runtime.h>
/*!
  \file printer_oned_multifluid.cu
  \brief Kernel to output 1D multifluid solution used by the PRINTER class
  \copyright Copyright (C) 2012-2015, Regents of the University of Michigan
  \license
  \author Marc T. Henry de Frahan <marchdf@umich.edu>, Computational Flow Physics Laboratory, University of Michigan
  \ingroup printer
*/
#ifdef ONED
#ifdef MULTIFLUID
#include <printer_kernels.h>

//==========================================================================
//
// Kernel definitions
//
//==========================================================================

//==========================================================================
arch_global void formater(int N_s, int N_E, scalar* U, scalar* output, bool inverse){
  /*!
    \brief Format solution kernel.
    \param[in] N_s number of nodes per element
    \param[in] N_E number of elements
    \param[in] U solution to format to output
    \param[out] output output solution array
    \param[in] inverse true if you want to copy from output to U instead
  */

  //
  // Copy from U to output
  //
  if(!inverse){
#ifdef USE_CPU
    for(int e = 0; e < N_E; e++){
      for(int i = 0; i < N_s; i++){
#elif USE_GPU
    int e = blockIdx.x*blkE+threadIdx.z;
    if (e < N_E){
      int i = threadIdx.x;{
#endif

	// Separate the fields
	scalar rho = U[(e*N_F+0)*N_s+i];
	scalar ux  = U[(e*N_F+1)*N_s+i]/rho;
	scalar et  = U[(e*N_F+2)*N_s+i];
#ifdef GAMCONS
	scalar gamma = 1+rho/U[(e*N_F+3)*N_s+i];
#elif  GAMNCON
	scalar gamma = 1+1.0/U[(e*N_F+3)*N_s+i];
#endif

	output[(e*N_F+0)*N_s+i] = rho;
	output[(e*N_F+1)*N_s+i] = ux;
	output[(e*N_F+2)*N_s+i] = gamma;
	output[(e*N_F+3)*N_s+i] = (gamma-1)*(et - 0.5*ux*ux*rho);
      
	// Mass fractions
#include "loopstart.h"
#define LOOP_END N_Y
#define MACRO(x) output[(e*N_F+4+x)*N_s+i] = U[(e*N_F+4+x)*N_s+i]/rho;
#include "loop.h"
      } // loop on nodes
    }  // loop on elements
  } // if inverse

  //
  // Copy from output to U
  //
  else {
#ifdef USE_CPU
    for(int e = 0; e < N_E; e++){
      for(int i = 0; i < N_s; i++){
#elif USE_GPU
    int e = blockIdx.x*blkE+threadIdx.z;
    if (e < N_E){
      int i = threadIdx.x;{
#endif

	// Get fields from output
	scalar rho   = output[(e*N_F+0)*N_s+i];
	scalar ux    = output[(e*N_F+1)*N_s+i];
	scalar gamma = output[(e*N_F+2)*N_s+i];
	scalar p     = output[(e*N_F+3)*N_s+i];

	U[(e*N_F+0)*N_s+i] = rho;
	U[(e*N_F+1)*N_s+i] = rho*ux;
	U[(e*N_F+2)*N_s+i] = p/(gamma-1) + 0.5*rho*ux*ux;
#ifdef GAMCONS
	U[(e*N_F+3)*N_s+i] = rho/(gamma-1);
#elif  GAMNCON
	U[(e*N_F+3)*N_s+i] = 1.0/(gamma-1);
#endif
	
	// Mass fractions
#include "loopstart.h"
#define LOOP_END N_Y
#define MACRO(x) U[(e*N_F+4+x)*N_s+i] = rho*output[(e*N_F+4+x)*N_s+i];
#include "loop.h"
      } // loop on nodes
    }  // loop on elements
  } // if inverse
}


//==========================================================================
//
//  Host C functions
//
//==========================================================================

extern "C"
void Lformater(int N_s, int N_E, scalar* U, scalar* output, bool inverse){
  /*!
    \brief Host C function to lauch format kernel.
    \param[in] N_s number of nodes per element
    \param[in] N_E number of elements
    \param[in] U solution to format to output
    \param[out] output output solution array
    \param[in] inverse true if you want to copy from output to U instead (default false)
    \section Description
    In GPU mode, launches N_E/blkE blocks of N_s x 1 x blkE
    threads. blkE controls the number of elements to set on each block
  */
#ifdef USE_GPU
  int div = N_E/blkE;
  int mod = 0;
  if (N_E%blkE != 0) mod = 1;
  dim3 dimBlock(N_s,1,blkE);
  dim3 dimGrid(div+mod,1);
#endif

  formater arch_args (N_s, N_E, U, output, inverse);
};
#endif
#endif
