#include "hip/hip_runtime.h"
/*!
  \file limiting_kernels.cu
  \brief Kernels used by the Limiting class
  \author Marc T. Henry de Frahan <marchdf@gmail.com>
  \ingroup limiting
*/
#include <limiting_kernels.h>
#include <stdio.h>

//==========================================================================
//
// Internal prototype function definitions
//
//==========================================================================

arch_device void getTaylorDerivative(int order, int N_s, scalar* T, int mx, int my, int* DxIdx, int* DyIdx, scalar* ddT);
arch_device scalar CellAvg(int N_G, int ioff, scalar* weight, scalar refArea, scalar* powers, int N_s, scalar* T);

arch_device inline int signum(scalar val){return val>0? 1 : (val<0? -1 : 0);}
arch_device scalar minmod(scalar a, scalar b);
arch_device inline scalar minabs(scalar* c, int n);
arch_device scalar minmod(scalar* c, int n);
arch_device scalar minmod2(scalar* c, int n);
arch_device int lim_factorial(int n);
arch_device void limit_monomial(int N, scalar* AL, scalar* AC, scalar* AR, scalar* Alim);
arch_device int binomial_coefficient(int n, int k);
arch_device void gemm(int M, int N, int K, scalar* A, scalar* B, scalar*C);
arch_device void gemm3(int M, int N, int K, scalar* A, scalar* B1, scalar*C1, scalar* B2, scalar*C2, scalar* B3, scalar*C3);
arch_device void gemv(int M, int N, scalar* A, scalar* B, scalar*C);
arch_device void gemv3(int M, int N, scalar* A, scalar* B1, scalar*C1, scalar* B2, scalar*C2, scalar* B3, scalar*C3);
arch_device inline scalar integrate_monomial_derivative(int k, int n);
arch_device inline scalar integrate_monomial_derivative_bounds(int k, int n, scalar a, scalar b);
arch_device void average_monomial(int N, scalar* A, scalar* Alim);

arch_device void set2average(int N_s, int N, int N_s1D, int slicenum, scalar* L2M, scalar* M2L, scalar* tmp, scalar* U, scalar* UMonoLim);
arch_device void HR(int N_s, int N, int N_s1D, int slicenum, scalar* L2M, scalar* M2L, scalar* tmp, scalar* UL, scalar* UC, scalar* UR, scalar* UMonoLim);
arch_device void pressure(int N_s, scalar* rho, scalar* rhou, scalar* rhov, scalar* E, scalar* gamma, scalar* beta, scalar* p);
arch_device void kinetic_energy(int N_s, scalar* L2M, scalar* rho, scalar* rhou, scalar* rhov, scalar* tmp, scalar* K);
arch_device void internal_energy(int N_s1D, int slicenum, scalar* p, scalar* g, scalar* b, scalar* rhoe);
arch_device void reconstruct_total_energy(int N_s, int N_s1D, int slicenum, scalar* L2M, scalar* M2L, scalar* rhoeLim, scalar* KLim, scalar* tmp, scalar* E);

//==========================================================================
//
// Kernel definitions
//
//==========================================================================

//==========================================================================
arch_global void stridedcopy(int numblocks, int blocklen, int strideA, int strideB, int offsetA, int offsetB, scalar* A, scalar* B){
  /*!
    \brief Strided copy of array A (length>= numblocks*strideA) to array B (length>= numblocks*strideB)
    \param[in] numblocks number of blocks to copy from A to B
    \param[in] blocklen number of elements in each block
    \param[in] strideA number of elements between start of each block in A
    \param[in] strideB number of elements between start of each block in B
    \param[in] offsetA number of elements to skip at start of A
    \param[in] offsetB number of elements to skip at start of B
    \param[in] A source array
    \param[out] B destination array
    \section Description    
    Modeled on MPI_Type_Vector
    
    You can test with this segment of code:
    scalar* a = new scalar[18];
    scalar* b = new scalar[6];
    for(int i=0; i<18; i++){a[i] = i;printf("%i %f\n",i,a[i]);}
    scalar* d_a;
    scalar* d_b;
    hipMalloc((void**) &d_a,18*sizeof(scalar));
    hipMalloc((void**) &d_b,6*sizeof(scalar));
    hipMemcpy(d_a, a, 18*sizeof(scalar), hipMemcpyHostToDevice);
    Lstridedcopy(2,3,9,3,0,0,d_a,d_b);
    hipMemcpy(b, d_b, 6*sizeof(scalar), hipMemcpyDeviceToHost);
    for(int i=0; i<6; i++){printf("%i: %f\n",i,b[i]);}
    delete[] a;
    delete[] b;
    exit(0);
  */

  
#ifdef USE_CPU

  int indA=offsetA,indB=offsetB;

  // Loop on number of blocks
  for(int i = 0; i < numblocks; i++){

    // Copy each block into B
    for(int j = 0; j < blocklen; j++){
      B[indB+j] = A[indA+j];
    }

    indA = indA+strideA;
    indB = indB+strideB;
  }

#elif USE_GPU
  int i = blockIdx.x*blkE+threadIdx.z;
  int indA=offsetA+i*strideA;
  int indB=offsetB+i*strideB;
  if (i < numblocks){
    int j = threadIdx.x;
    B[indB+j] = A[indA+j];
  }    
#endif  
}

//==========================================================================
arch_global void reconstruct_energy(int N_s, int N_E, int slicenum, scalar* rhoeLim, scalar* KLim, scalar* EMono, scalar* ELim){
  /*!
    \brief Reconstruct the energy monomial coefficients
    \param[in] N_s number of nodes per element
    \param[in] N_E number of elements
    \param[in] slicenum to decompose higher dimensional problem into 1D slices
    \param[in] rhoeLim limited monomial internal energy
    \param[in] KLim limited monomial kinetic energy
    \param[in] EMono monomial total energy
    \param[out] ELim limited monomial total energy
    \section Description
    Reconstruct the energy monomial coefficients using the internal
    and kinetic energy monomial coefficients. Apply a correction to the
    zeroth coefficients so that the method is conservative.
  */
#ifdef USE_CPU
  for(int e = 0; e < N_E; e++){
    for(int slice = 0; slice < slicenum; slice++){
#elif USE_GPU
  int e = blockIdx.x*blkE+threadIdx.z;
  if (e < N_E){
    int slice = threadIdx.x;
#endif

    int idx=0;
    int idx0=e*N_s*slicenum+slice*N_s+0;
    
    // Start at idx 1 because we will do the zeroth coefficient separately
    for(int i = 1; i < N_s; i++){
      idx = e*N_s*slicenum+slice*N_s+i;
      ELim[idx] = rhoeLim[idx]+KLim[idx];
    }

    // Correct the zeroth coefficient to conserve energy
    scalar E0 = EMono[idx0];
    for(int k = 2; k<N_s; k+=2){
      idx = e*N_s*slicenum+slice*N_s+k;
      E0 -= 1.0/((scalar)lim_factorial(k+1)) * (ELim[idx]-EMono[idx]);
    }
    ELim[idx0] = E0;

#ifdef USE_CPU
  }// for on slice
#endif
  }
}


//==========================================================================
arch_global void internal_energy_multifluid(int N_s, int N_E, int slicenum, scalar* p, scalar* g, scalar* rhoe){
  /*!
    \brief Reconstruct the energy monomial coefficients
    \param[in] N_s number of nodes per element
    \param[in] N_E number of elements
    \param[in] slicenum to decompose higher dimensional problem into 1D slices
    \param[in] p monomial pressure solution
    \param[in] g monomial 1/(gamma-1) solution
    \param[out] rhoe monomial internal energy
    \section Description
    Reconstruct the monomial internal energy coefficients using the
    pressure and 1/gamma-1 coefficients so that the pressure remains
    non-oscillatory
  */
#ifdef USE_CPU
  for(int e = 0; e < N_E; e++){
    for(int slice = 0; slice < slicenum; slice++){
      for(int i = 0; i < N_s; i++){
#elif USE_GPU
  int e = blockIdx.x*blkE+threadIdx.z;
  if (e < N_E){
    int i = threadIdx.x;
    int slice = threadIdx.y;
#endif

    //printf("==== m = %i\n",i);
    scalar I = 0;
    for(int k=0; k<i+1; k++){
      // could prob do this faster if I brought p and g as a shared array
      //printf("(m,k)=(%i,%i)=%i, m-k=%i, k=%i\n",i,k,binomial_coefficient(i,k),i-k,k);
      I += (scalar)binomial_coefficient(i,k) * p[e*N_s*slicenum+slice*N_s+i-k] * g[e*N_s*slicenum+slice*N_s+k];
    }
    rhoe[e*N_s*slicenum+slice*N_s+i] = I;

#ifdef USE_CPU
    }
  }
#endif
  }
}

//==========================================================================
arch_global void internal_energy_stiffened(int N_s, int N_E, int slicenum, scalar* p, scalar* g, scalar* b, scalar* rhoe){
  /*!
    \brief Reconstruct the energy monomial coefficients
    \param[in] N_s number of nodes per element
    \param[in] N_E number of elements
    \param[in] slicenum to decompose higher dimensional problem into 1D slices
    \param[in] p monomial pressure solution
    \param[in] g monomial 1/(gamma-1) solution
    \param[in] b monomial gamma*pinf/(gamma-1) solution
    \param[out] rhoe monomial internal energy
    \section Description
    Reconstruct the monomial internal energy coefficients using the
    pressure, 1/gamma-1, and gamma*pinf/(gamma-1) coefficients so that
    the pressure remains non-oscillatory
  */
#ifdef USE_CPU
  for(int e = 0; e < N_E; e++){
    for(int slice = 0; slice < slicenum; slice++){
      for(int i = 0; i < N_s; i++){
#elif USE_GPU
  int e = blockIdx.x*blkE+threadIdx.z;
  if (e < N_E){
    int i = threadIdx.x;
    int slice = threadIdx.y;
#endif

    //printf("==== m = %i\n",i);
    scalar I = 0;
    for(int k=0; k<i+1; k++){
      // could prob do this faster if I brought p and g as a shared array
      //printf("(m,k)=(%i,%i)=%i, m-k=%i, k=%i\n",i,k,binomial_coefficient(i,k),i-k,k);
      I += (scalar)binomial_coefficient(i,k) * p[e*N_s*slicenum+slice*N_s+i-k] * g[e*N_s*slicenum+slice*N_s+k];
    }
    rhoe[e*N_s*slicenum+slice*N_s+i] = I + b[e*N_s*slicenum+slice*N_s+i];

#ifdef USE_CPU
  }
  }
#endif
  }
}

//==========================================================================
arch_global void hrl1D(int N_s, int N_E, int Nfields, int N_N, int slicenum, int* neighbors, int offxy, scalar* A, scalar* Alim){
  /*!
    \brief HR limiting function (assumes 1D decomposition)
    \param[in] N_s number of nodes per element
    \param[in] N_E number of elements
    \param[in] Nfields number of fields to operate on (eg. one field instead of N_F)
    \param[in] N_N number of neighbors per element
    \param[in] slicenum to decompose higher dimensional problem into 1D slices
    \param[in] neighbors array containing an element's neighbors
    \param[in] offxy offset if limiting in x or y
    \param[in] A solution to limit (monomial form)
    \param[out] Alim limited solution (monomial form)
  */
#ifdef USE_CPU
  for(int e = 0; e < N_E; e++){
    for(int slice = 0; slice < slicenum; slice++){
      for(int fc = 0; fc < Nfields; fc++){
#elif USE_GPU
  int blk = threadIdx.z; 
  int e = blockIdx.x*blkE+blk;
  if (e < N_E){
    int slice= threadIdx.x;
    int fc= threadIdx.y;
#endif  

	int N = N_s-1;

	int left  = neighbors[e*N_N+offxy+0];
	int right = neighbors[e*N_N+offxy+1];

	// Check to see if we are at a boundary
	int physical = 0;
	if (left  < 0){physical = -left;}
	if (right < 0){physical = -right;}

	// gravity field: leave data unchanged. Not good for shocks
	if (physical==4){} 
	
	// Zero-gradient and reflective BC: average in cell, slopes to 0
	else if ((physical==2)||(physical==3)){
	  int idx = (e*Nfields+fc)*N_s*slicenum+slice*N_s;
	  average_monomial(N, &A[idx], &Alim[idx]);
	}
	
	//Otherwise do the full limiting
	else{
	  int idxL = (left *Nfields+fc)*N_s*slicenum+slice*N_s;
	  int idxC = (e    *Nfields+fc)*N_s*slicenum+slice*N_s;
	  int idxR = (right*Nfields+fc)*N_s*slicenum+slice*N_s;
	  limit_monomial(N,&A[idxL],&A[idxC],&A[idxR],&Alim[idxC]);      
	} // end if on physicals

#ifdef USE_CPU
      }
    }
#endif
  }
}

//==========================================================================
arch_global void hri1D(int N_s, int N_E, int N_N, int* neighbors, int N_s1D, int slicenum, int offxy, scalar* Lag2Mono, scalar* Mono2Lag, int* sensors, scalar* U, scalar* Unew){
  /*!
    \brief HR limiting function for individual elements (assumes 1D decomposition)
    \param[in] N_s number of nodes per element
    \param[in] N_E number of elements
    \param[in] N_N number of neighbors per element
    \param[in] neighbors array containing an element's neighbors
    \param[in] N_s1D number of nodes in a slice (1D element)
    \param[in] slicenum number of slices (in 2D N_s1D = slicenum)
    \param[in] offxy offset if limiting in x or y
    \param[in] sensors array of sensors
    \param[in] U solution to limit (Lagrange form)
    \param[out] Unew limited solution (only some may be limited bc of sensor)
    Unew was necessary because you need to wait until all the elements
    have been limited before updating the solution.
  */ 

  int N = N_s1D-1; // polynomial order

#ifdef USE_CPU
  scalar* share = new scalar[7*N_s];
  for(int e=0; e<N_E; e++){
    int sen = sensors[e];
    if (sen != 0){
      for(int fc=0; fc<N_F; fc++){
	int cnt = 0;
	scalar* UL  = &share[cnt]; cnt += N_s;
	scalar* UC  = &share[cnt]; cnt += N_s;
	scalar* UR  = &share[cnt]; cnt += N_s;
	scalar* tmp = &share[cnt]; //cnt += 4*N_s;
	scalar* L2M = Lag2Mono;
	scalar* M2L = Mono2Lag;

#elif USE_GPU
  int e = blockIdx.x;{
    int sen = sensors[e];
    if (sen != 0){
      int fc= threadIdx.y;{
	extern __shared__ scalar share[];
      
	// offset wrt other shared data
	int cnt = fc*(2*N_s*N_s + 7*N_s);
	scalar* L2M = &share[cnt]; cnt += N_s*N_s;
	scalar* M2L = &share[cnt]; cnt += N_s*N_s;
	scalar* UL  = &share[cnt]; cnt += N_s;
	scalar* UC  = &share[cnt]; cnt += N_s;
	scalar* UR  = &share[cnt]; cnt += N_s;
	scalar* tmp = &share[cnt]; //cnt += 4*N_s;
	// Copy some data to shared memory
	for(int k=0;k<N_s*N_s;k++){L2M[k] = Lag2Mono[k];}
	for(int k=0;k<N_s*N_s;k++){M2L[k] = Mono2Lag[k];}
#endif
    
	// Neighbors
	int left  = neighbors[e*N_N+offxy+0];
	int right = neighbors[e*N_N+offxy+1];
    
	// Check to see if we are at a boundary
	int physical = 0;
	if (left  < 0){physical = -left;}
	if (right < 0){physical = -right;}

	// Copy some data to shared memory
	for(int i=0;i<N_s;i++){UC[i]=U[(e*N_F+fc)*N_s+i];}
    
	// gravity field: leave data unchanged. Not good for shocks
	if (physical==4){} 
    
	// Zero-gradient and reflective BC: average in cell, slopes to 0
	else if ((physical==2)||(physical==3)){
	  set2average(N_s,N,N_s1D,slicenum,L2M,M2L,tmp,UC,NULL);
	}
  
	//Otherwise do the full limiting
	else{
	  for(int i=0;i<N_s;i++){UL[i]=U[(left *N_F+fc)*N_s+i];}
	  for(int i=0;i<N_s;i++){UR[i]=U[(right*N_F+fc)*N_s+i];}
	  HR(N_s,N,N_s1D,slicenum,L2M,M2L,tmp,UL,UC,UR,NULL);
	} // end if on physicals
  
	// Copy solution back to main memory
	for(int i=0;i<N_s;i++){Unew[(e*N_F+fc)*N_s+i] = UC[i];}
	
	L2M = NULL; M2L = NULL; UL=NULL; UC = NULL; UR = NULL; tmp = NULL;
      } // loop on fields
    } // if on sensor
  } // loop on elements

#ifdef USE_CPU
  delete[] share;
#endif
}


//==========================================================================
arch_global void m2i1D(int N_s, int N_E, int N_N, int* neighbors, int N_s1D, int slicenum, int offxy, scalar* Lag2Mono, scalar* Mono2Lag, int* sensors, scalar* U, scalar* Unew){
  /*!
    \brief Modified limiting function for individual elements (assumes 1D decomposition)
    \param[in] N_s number of nodes per element
    \param[in] N_E number of elements
    \param[in] N_N number of neighbors per element
    \param[in] neighbors array containing an element's neighbors
    \param[in] N_s1D number of nodes in a slice (1D element)
    \param[in] slicenum number of slices (in 2D N_s1D = slicenum)
    \param[in] offxy offset if limiting in x or y
    \param[in] sensors array of sensors
    \param[in] U solution to limit (Lagrange form)
    \param[out] Unew limited solution (only some may be limited bc of sensor)
    Unew was necessary because you need to wait until all the elements
    have been limited before updating the solution.
  */
 
  int size_share = N_F*3*N_s +  // room for L/C/R for all the fields
    N_s +  // for gammaLim
#ifdef STIFFENED
    N_s + // for betaLim
#endif 
    N_s +  // for pressureLim
    3*N_s + // for E L/C/R
    N_s + // for KLim
    N_s + // for rhoeLim
    4*N_s; // for tmp

  int N = N_s1D-1; // polynomial order

#ifdef USE_CPU  
  scalar* share = new scalar[size_share];
  for(int e=0; e<N_E; e++){
    int cnt = 0;
    int sen = sensors[e];
    if (sen != 0){
      scalar* L2M = Lag2Mono;
      scalar* M2L = Mono2Lag;
#elif USE_GPU
  extern __shared__ scalar share[];
  int e = blockIdx.x;{
    int cnt = 0;
    int sen = sensors[e];
    if (sen != 0){
      scalar* L2M = &share[cnt]; cnt+=N_s*N_s;
      scalar* M2L = &share[cnt]; cnt+=N_s*N_s;
      for(int k=0;k<N_s*N_s;k++){L2M[k] = Lag2Mono[k];}
      for(int k=0;k<N_s*N_s;k++){M2L[k] = Mono2Lag[k];}
#endif

      // Initialize pointers
      scalar* rhoL = &share[cnt]; cnt += N_s;
      scalar* rhoC = &share[cnt]; cnt += N_s;
      scalar* rhoR = &share[cnt]; cnt += N_s;
      scalar* rhouL = &share[cnt]; cnt += N_s;
      scalar* rhouC = &share[cnt]; cnt += N_s;
      scalar* rhouR = &share[cnt]; cnt += N_s;
#ifdef TWOD
      scalar* rhovL = &share[cnt]; cnt += N_s;
      scalar* rhovC = &share[cnt]; cnt += N_s;
      scalar* rhovR = &share[cnt]; cnt += N_s;
#else
      scalar* rhovL=NULL, *rhovC=NULL, *rhovR=NULL;
#endif
      scalar* EL = &share[cnt]; cnt += N_s;
      scalar* EC = &share[cnt]; cnt += N_s;
      scalar* ER = &share[cnt]; cnt += N_s;
      scalar* gammaL = &share[cnt]; cnt += N_s;
      scalar* gammaC = &share[cnt]; cnt += N_s;
      scalar* gammaR = &share[cnt]; cnt += N_s;
      scalar* gammaLim = &share[cnt]; cnt += N_s;
#ifdef STIFFENED
      scalar* betaL = &share[cnt]; cnt += N_s;
      scalar* betaC = &share[cnt]; cnt += N_s;
      scalar* betaR = &share[cnt]; cnt += N_s;
      scalar* betaLim = &share[cnt]; cnt += N_s;
#else
      scalar* betaL=NULL, *betaC = NULL, *betaR = NULL, *betaLim = NULL;
#endif
      scalar* pressureL = &share[cnt]; cnt += N_s;
      scalar* pressureC = &share[cnt]; cnt += N_s;
      scalar* pressureR = &share[cnt]; cnt += N_s;
      scalar* pressureLim = &share[cnt]; cnt += N_s;
      scalar* KLim = &share[cnt]; cnt += N_s;
      scalar* rhoeLim = &share[cnt]; cnt += N_s;
      scalar* tmp = &share[cnt]; cnt+= 4*N_s;

      // Mass fractions
#include "loopstart.h"
#define LOOP_END N_Y
#define MACRO(x) scalar* YL(x) = &share[cnt]; cnt+=N_s; \
      scalar* YC(x) = &share[cnt]; cnt+=N_s;		\
      scalar* YR(x) = &share[cnt]; cnt+=N_s;
#include "loop.h"    

      // Neighbors
      int left  = neighbors[e*N_N+offxy+0];
      int right = neighbors[e*N_N+offxy+1];
    
      // Check to see if we are at a boundary
      int physical = 0;
      if (left  < 0){physical = -left;}
      if (right < 0){physical = -right;}

      // Copy some data to shared memory
      int fcnt=0;
      for(int i=0; i<N_s; i++){rhoC[i]   = U[(e*N_F+fcnt)*N_s+i];} fcnt++;
      for(int i=0; i<N_s; i++){rhouC[i]  = U[(e*N_F+fcnt)*N_s+i];} fcnt++;
#ifdef TWOD
      for(int i=0; i<N_s; i++){rhovC[i]  = U[(e*N_F+fcnt)*N_s+i];} fcnt++;
#endif
      for(int i=0; i<N_s; i++){EC[i]     = U[(e*N_F+fcnt)*N_s+i];} fcnt++;
      for(int i=0; i<N_s; i++){gammaC[i] = U[(e*N_F+fcnt)*N_s+i];} fcnt++;
#ifdef STIFFENED
      for(int i=0; i<N_s; i++){betaC[i]  = U[(e*N_F+fcnt)*N_s+i];} fcnt++;
#endif 
#include "loopstart.h"
#define LOOP_END N_Y
#define MACRO(x)  for(int i=0; i<N_s; i++){YC(x)[i] = U[(e*N_F+fcnt)*N_s+i];} fcnt++;
#include "loop.h"    

      // Get the pressure
      pressure(N_s,rhoC,rhouC,rhovC,EC,gammaC,betaC,pressureC);

      // gravity field: leave data unchanged. Not good for shocks
      if (physical==4){} 
  
      // Zero-gradient and reflective BC: average in cell, slopes to 0
      else if ((physical==2)||(physical==3)){
	set2average(N_s,N,N_s1D,slicenum,L2M,M2L,tmp,rhoC,NULL);
	set2average(N_s,N,N_s1D,slicenum,L2M,M2L,tmp,rhouC,NULL);
#ifdef TWOD
	set2average(N_s,N,N_s1D,slicenum,L2M,M2L,tmp,rhovC,NULL);
#endif
	set2average(N_s,N,N_s1D,slicenum,L2M,M2L,tmp,gammaC,gammaLim);
#ifdef STIFFENED
	set2average(N_s,N,N_s1D,slicenum,L2M,M2L,tmp,betaC,betaLim);
#endif
	set2average(N_s,N,N_s1D,slicenum,L2M,M2L,tmp,pressureC,pressureLim);
#include "loopstart.h"
#define LOOP_END N_Y
#define MACRO(x) set2average(N_s,N,N_s1D,slicenum,L2M,M2L,tmp,YC(x),NULL);
#include "loop.h"    

	// Reconstruct the energies
	// Kinetic (with limited Lagrange rho,rhou,rhov)
	kinetic_energy(N_s, L2M, rhoC, rhouC, rhovC, tmp, KLim);

	// Internal (with limited monomial p, gamma, beta)
	internal_energy(N_s1D, slicenum, pressureLim, gammaLim, betaLim, rhoeLim);

	// Total
	reconstruct_total_energy(N_s, N_s1D, slicenum, L2M, M2L, rhoeLim, KLim, tmp, EC);
      }

      //Otherwise do the full limiting
      else{
	// Copy left/right data to shared memory
	fcnt=0;
	for(int i=0; i<N_s; i++){rhoL[i]   = U[(left *N_F+fcnt)*N_s+i];} 
	for(int i=0; i<N_s; i++){rhoR[i]   = U[(right*N_F+fcnt)*N_s+i];} fcnt++;
	for(int i=0; i<N_s; i++){rhouL[i]  = U[(left *N_F+fcnt)*N_s+i];} 
	for(int i=0; i<N_s; i++){rhouR[i]  = U[(right*N_F+fcnt)*N_s+i];} fcnt++;
#ifdef TWOD
	for(int i=0; i<N_s; i++){rhovL[i]  = U[(left *N_F+fcnt)*N_s+i];} 
	for(int i=0; i<N_s; i++){rhovR[i]  = U[(right*N_F+fcnt)*N_s+i];} fcnt++;
#endif
	for(int i=0; i<N_s; i++){EL[i]     = U[(left *N_F+fcnt)*N_s+i];} 
	for(int i=0; i<N_s; i++){ER[i]     = U[(right*N_F+fcnt)*N_s+i];} fcnt++;
	for(int i=0; i<N_s; i++){gammaL[i] = U[(left *N_F+fcnt)*N_s+i];} 
	for(int i=0; i<N_s; i++){gammaR[i] = U[(right*N_F+fcnt)*N_s+i];} fcnt++;
#ifdef STIFFENED
	for(int i=0; i<N_s; i++){betaL[i]  = U[(left *N_F+fcnt)*N_s+i];} 
	for(int i=0; i<N_s; i++){betaR[i]  = U[(right*N_F+fcnt)*N_s+i];} fcnt++;
#endif 
#include "loopstart.h"
#define LOOP_END N_Y
#define MACRO(x)  for(int i=0; i<N_s; i++){YL(x)[i]  = U[(left *N_F+fcnt)*N_s+i];} \
	for(int i=0; i<N_s; i++){YR(x)[i]  = U[(right*N_F+fcnt)*N_s+i];} fcnt++;
#include "loop.h"    

	// Get the pressure
	pressure(N_s,rhoL,rhouL,rhovL,EL,gammaL,betaL,pressureL);
	pressure(N_s,rhoR,rhouR,rhovR,ER,gammaR,betaR,pressureR);

	// Limit
	HR(N_s,N,N_s1D,slicenum,L2M,M2L,tmp,rhoL,rhoC,rhoR,NULL);
	HR(N_s,N,N_s1D,slicenum,L2M,M2L,tmp,rhouL,rhouC,rhouR,NULL);
#ifdef TWOD
	HR(N_s,N,N_s1D,slicenum,L2M,M2L,tmp,rhovL,rhovC,rhovR,NULL);
#endif
	HR(N_s,N,N_s1D,slicenum,L2M,M2L,tmp,gammaL,gammaC,gammaR,gammaLim);
#ifdef STIFFENED
	HR(N_s,N,N_s1D,slicenum,L2M,M2L,tmp,betaL,betaC,betaR,betaLim);
#endif
	HR(N_s,N,N_s1D,slicenum,L2M,M2L,tmp,pressureL,pressureC,pressureR,pressureLim);
#include "loopstart.h"
#define LOOP_END N_Y
#define MACRO(x) HR(N_s,N,N_s1D,slicenum,L2M,M2L,tmp,YL(x),YC(x),YR(x),NULL); 
#include "loop.h"    

	// Reconstruct the energies
	// Kinetic (with limited Lagrange rho,rhou,rhov)
	kinetic_energy(N_s, L2M, rhoC, rhouC, rhovC, tmp, KLim);

	// Internal (with limited monomial p, gamma, beta)
	internal_energy(N_s1D, slicenum, pressureLim, gammaLim, betaLim, rhoeLim);

	// Total
	reconstruct_total_energy(N_s, N_s1D, slicenum, L2M, M2L, rhoeLim, KLim, tmp, EC);
      } // end if on physicals
  
      // Copy solution back to main memory
      fcnt=0;
      for(int i=0; i<N_s; i++){Unew[(e*N_F+fcnt)*N_s+i] = rhoC[i];} fcnt++;
      for(int i=0; i<N_s; i++){Unew[(e*N_F+fcnt)*N_s+i] = rhouC[i];} fcnt++;
#ifdef TWOD		                         
      for(int i=0; i<N_s; i++){Unew[(e*N_F+fcnt)*N_s+i] = rhovC[i];} fcnt++;
#endif			                         
      for(int i=0; i<N_s; i++){Unew[(e*N_F+fcnt)*N_s+i] = EC[i];} fcnt++;
      for(int i=0; i<N_s; i++){Unew[(e*N_F+fcnt)*N_s+i] = gammaC[i];} fcnt++;
#ifdef STIFFENED	                         
      for(int i=0; i<N_s; i++){Unew[(e*N_F+fcnt)*N_s+i] = betaC[i];} fcnt++;
#endif 
#include "loopstart.h"
#define LOOP_END N_Y
#define MACRO(x)  for(int i=0; i<N_s; i++){Unew[(e*N_F+fcnt)*N_s+i] = YC(x)[i];} fcnt++;
#include "loop.h"    

      // Delete pointers
      rhoL=NULL; rhoC=NULL; rhoR=NULL; rhouL=NULL; rhouC=NULL; rhouR=NULL;
#ifdef TWOD
      rhovL=NULL; rhovC=NULL; rhovR=NULL;
#endif
      EL=NULL; EC=NULL; ER=NULL; gammaL=NULL; gammaC=NULL; gammaR=NULL; gammaLim=NULL;
#ifdef STIFFENED
      betaL=NULL; betaC=NULL; betaR=NULL; betaLim=NULL;
#endif
      pressureL=NULL; pressureC=NULL; pressureL=NULL; pressureLim=NULL;
      KLim=NULL; rhoeLim=NULL; tmp=NULL;
#include "loopstart.h"
#define LOOP_END N_Y
#define MACRO(x)  YL(x)=NULL; YC(x)=NULL; YR(x)=NULL; 
#include "loop.h"
    } // if on sensor
  } // loop on elements
#ifdef USE_CPU
  delete[] share;
#endif
  
}
  
//==========================================================================
arch_global void hrl2D(int N_s, int N_E, int N_G, int N_N, int order, scalar* XYZCen, scalar* powersXYZG, int* neighbors, int* TaylorDxIdx, int* TaylorDyIdx, scalar* weight, scalar refArea, scalar* A, scalar* Alim){
  /*!
    \brief Not used right now. HR limiting function fully 2D (eg. for triangles)
    \param[in] N_s number of nodes per element
    \param[in] N_E number of elements
    \param[in] N_G number of gaussian nodes per element
    \param[in] N_N number of neighbors per element
    \param[in] order DG order
    \param[in] XYZCen element centroid coordinates
    \param[in] powersXYZG powers of coordinates of gaussian nodes
    \param[in] neighbors array containing an element's neighbors
    \param[in] TaylorDxIdx indices of Taylor polynomial derivatives in x
    \param[in] TaylorDyIdx indices of Taylor polynomial derivatives in y
    \param[in] weight gaussian integration weights
    \param[in] refArea area of reference triangle
    \param[in] A solution to limit (monomial form)
    \param[out] Alim limited solution (monomial form)
  */
#ifdef USE_CPU
  for(int e = 0; e < N_E; e++){
    for(int fc = 0; fc < N_F; fc++){
#elif USE_GPU
  int e = blockIdx.x*blkE+threadIdx.z;
  if (e < N_E){
      int fc= threadIdx.y;
      extern __shared__ scalar c[];
#endif  

//       // Allocate resources locally
//       scalar* localU = new scalar[N_s*(N_N+1)];
//       scalar* neighPowers = new scalar[N_s*N_G*(N_N+1)]; // powers of X in the cell + neighbors
//       scalar* localPowers = new scalar[N_s*N_G*(N_N+1)]; // powers of X to eval cell poly in neighbors
//       scalar* localXYZCen = new scalar[D*(N_N+1)];
//       scalar* dT = new scalar[N_s]; for(int i=0;i<N_s;i++) dT[i]=0;
//       scalar* cx = new scalar[N_N]; // candidates wrt x to limit
//       scalar* cy = new scalar[N_N]; // candidates wrt y to limit
//       scalar* avgdU = new scalar[N_N+1];
//       scalar* avgR  = new scalar[N_N+1];
//       scalar* avgL  = new scalar[N_N+1];
//       scalar candA, candB, oldcandA;
	  
//       // Copy global to local resources (going to have to make these N_F times bigger for GPU)
//       int el = 0; // place holder for the elements
//       for(int nn = 0; nn < N_N+1; nn++){
//       	if(nn==0) el = e; // acting on the target element
//       	else      el = neighbors[e*N_N+nn-1]; // acting on his neighbors
//       	// Copy U
//       	for(int i = 0; i < N_s; i++) localU[nn*N_s+i] = A[(el*N_F+fc)*N_s+i];
//       	// Copy XYZ centroids
//       	//for(int alpha = 0; alpha < D; alpha ++){
//       	scalar cenx = 0, ceny=0;
//       	for(int g = 0; g < N_G; g++){
//       	  cenx += powersXYZG[((e*(N_N+1)+nn)*N_G+g)*N_s+1];
//       	  ceny += powersXYZG[((e*(N_N+1)+nn)*N_G+g)*N_s+2];
//       	}
//       	//localXYZCen[nn*D+alpha] = XYZCen[el*D+alpha];
//       	localXYZCen[nn*D+0] = cenx/N_G;
//       	localXYZCen[nn*D+1] = ceny/N_G;
//       	//}
//       	// Copy powers of XYZ
//       	for(int g = 0; g < N_G; g++){
//       	  for(int i = 0; i < N_s; i++){
//       	    neighPowers[(nn*N_G+g)*N_s+i] = powersXYZG[((el*(N_N+1)+0)*N_G+g)*N_s+i];
//       	    //if((nn==0)&&(e==0)&&(fc==0)) printf("%e ",neighPowers[(nn*N_G+g)*N_s+i]);
//       	    localPowers[(nn*N_G+g)*N_s+i] = powersXYZG[((e*(N_N+1)+nn)*N_G+g)*N_s+i];
//       	  }
//       	  //if((nn==0)&&(e==0)&&(fc==0)) printf("\n");
//       	}
//       }
      
//       // Loop on derivatives
//       for(int m = order; m > 0; m--){
//       	for(int k = 0; k < m; k++){ // loop on combinations of (m-1) order derivatives
//       	  int mx = m-1-k;
//       	  int my = k;
	  
//       	  for(int nn = 0; nn < N_N+1; nn++){ 
//       	    if(nn==0) el = e; // acting on the target element
//       	    else      el = neighbors[e*N_N+nn-1]; // acting on his neighbors

//       	    // Calculate the cell averages of the target polynomial and neighbor polynomials
//       	    for(int i=0;i<N_s;i++) dT[i]=0;
//       	    getTaylorDerivative(order, N_s, &localU[nn*N_s], mx, my, TaylorDxIdx, TaylorDyIdx, dT);
//       	    // if((e==8)&&(fc==0)){
//       	    //   printf("In element %i\n",el);
//       	    //   for(int i=0;i<N_s;i++){printf("    T[%i]=%f ",i,dT[i]);}
//       	    //   printf("\n");
//       	    // }
//       	    avgdU[nn] = CellAvg(N_G, 0, weight, refArea, &neighPowers[nn*N_G*N_s], N_s, dT);
	      
//       	    // Calculate the cell average of the target remainder (with limited coeffs) on element and neighbors
//       	    for(int i=0;i<N_s;i++) dT[i]=0;
//       	    //getTaylorDerivative(order, N_s, &localU[nn*N_s], mx, my, TaylorDxIdx, TaylorDyIdx, dT);
//       	    getTaylorDerivative(order, N_s, &Alim[(e*N_F+fc)*N_s], mx, my, TaylorDxIdx, TaylorDyIdx, dT);
//       	    //if((e==8)&&(fc==0)) for(int i=0;i<N_s;i++) printf("    T[%i]=%f ",i,dT[i]);
//       	    avgR[nn] = CellAvg(N_G, 3, weight, refArea, &localPowers[nn*N_G*N_s], N_s,dT);
      
//       	    // Estimate the cell averages
//       	    avgL[nn] = avgdU[nn] - avgR[nn];

//       	    //if((e==8)&&(fc==0))printf("avgdU=%f avgR=%f avgL=%f\n",avgdU[nn]*(0.5*0.44444),avgR[nn]*(0.5*0.4444444),avgL[nn]*(0.5*0.4444444));
//       	  }

//       	  // store in the coefficient vectors
//       	  for(int nn = 1; nn < N_N+1; nn++){ // loop on element + neighbors
//       	    cx[nn-1] = (avgL[nn] - avgL[0]) / (localXYZCen[nn*D+0]-localXYZCen[0*D+0]);
//       	    cy[nn-1] = (avgL[nn] - avgL[0]) / (localXYZCen[nn*D+1]-localXYZCen[0*D+1]);
//       	    //if((e==8)&&(fc==0)) printf("avgL(%i)=%f, avgL(0)=%f, dL=%f, dx=%f, dy=%f\n", nn,avgL[nn],avgL[0], avgL[nn] - avgL[0], (localXYZCen[nn*D+0]-localXYZCen[0*D+0]), (localXYZCen[nn*D+1]-localXYZCen[0*D+1]));
//       	  }

//       	  //Get the canditate coefficients
//       	  candA = minmod(cx,N_N);
//       	  candB = minmod(cy,N_N);
//       	  if     (k==0)   Alim[(e*N_F+fc)*N_s+m*(m+1)/2+k]   = candA;
//       	  else if(k==m-1){
//       	    Alim[(e*N_F+fc)*N_s+m*(m+1)/2+k]   = minmod(candA,oldcandA);
//       	    Alim[(e*N_F+fc)*N_s+m*(m+1)/2+k+1] = candB;
//       	  }
//       	  else Alim[(e*N_F+fc)*N_s+m*(m+1)/2+k]= minmod(candA,oldcandA);
//       	  oldcandA = candB;
//       	  //if((e==8)&&(fc==0))printf("candA=%f, candB=%f, oldcandA=%f\n",candA,candB,oldcandA);
	  
//       	  // Cell average invariance
//       	  if(m==1) Alim[(e*N_F+fc)*N_s+0] = avgL[0];

//       	} // loop on combinations
//       } // loop on m
      
//       delete[] localU;
//       delete[] neighPowers;
//       delete[] localPowers;
//       delete[] localXYZCen;
//       delete[] dT;
//       delete[] cx;
//       delete[] cy;
//       delete[] avgdU;
//       delete[] avgR;
//       delete[] avgL;
 
#ifdef USE_CPU
    }
#endif
  }
}

//==========================================================================
arch_global void ChangeBasis(int size1, int size2, int N_E, scalar* Transform, scalar* U, scalar* Unew){
  /*!
    \brief Basis transformation (manual). Do not use this. Use BLAS
    \param[in] size1 number of rows of tranform
    \param[in] size2 number of columns of tranform
    \param[in] N_E number of elements
    \param[in] Transform Basis transform matrix (per element)
    \param[in] U solution to transform
    \param[out] Unew transformed solution
  */
    
#ifdef USE_CPU
  for(int e = 0; e < N_E; e++){
    for(int i = 0; i < size1; i++){
      for(int fc = 0; fc < N_F; fc++){
#elif USE_GPU
  int e = blockIdx.x*blkE+threadIdx.z;
  if (e < N_E){
    int i = threadIdx.x;
    int fc= threadIdx.y;
#endif

  scalar sol = 0.0;
	
  for(int ii = 0; ii < size2; ii++){
    sol += Transform[(e*size1+i)*size2+ii]*U[(e*N_F+fc)*size2+ii];
  }
  Unew[(e*N_F+fc)*size1+i] = sol;
  sol = 0.0;

#ifdef USE_CPU
      }
    }
#endif
  }
}

//==========================================================================
//
//  Host C functions
//
//==========================================================================
extern "C" 
void Lstridedcopy(int numblocks, int blocklen, int strideA, int strideB, int offsetA, int offsetB, scalar* A, scalar* B){
  /*!
    \brief Host C function to lauch stridedcopy kernel.
    \param[in] numblocks number of blocks to copy from A to B
    \param[in] blocklen number of elements in each block
    \param[in] strideA number of elements between start of each block in A
    \param[in] strideB number of elements between start of each block in B
    \param[in] offsetA number of elements to skip at start of A
    \param[in] offsetB number of elements to skip at start of B
    \param[in] A source array
    \param[out] B destination array
    \section Description
    In GPU mode, launches numblocks/blkE blocks of blocklen x 1 x blkE
    threads. blkE controls the number of elements to set on each block
  */
#ifdef USE_GPU
  int div = numblocks/blkE;
  int mod = 0;
  if (numblocks%blkE != 0) mod = 1;
  dim3 dimBlock(blocklen,1,blkE);
  dim3 dimGrid(div+mod,1);
#endif

  stridedcopy arch_args (numblocks, blocklen, strideA, strideB, offsetA, offsetB, A, B);
};

extern "C"
void Lreconstruct_energy(int N_s, int N_E, int slicenum, scalar* rhoeLim, scalar* KLim, scalar* EMono, scalar* ELim){
  /*!
    \brief Host C function to lauch reconstruct_energy kernel.
    \param[in] N_s number of nodes per element
    \param[in] N_E number of elements
    \param[in] slicenum to decompose higher dimensional problem into 1D slices
    \param[in] rhoeLim limited monomial internal energy
    \param[in] KLim limited monomial kinetic energy
    \param[in] EMono monomial total energy
    \param[out] ELim limited monomial total energy
    \section Description
    In GPU mode, launches N_E/blkE blocks of slicenum x 1 x blkE
    threads. blkE controls the number of elements to set on each block
  */
#ifdef USE_GPU
  int div = N_E/blkE;
  int mod = 0;
  if (N_E%blkE != 0) mod = 1;
  dim3 dimBlock(slicenum,1,blkE);
  dim3 dimGrid(div+mod,1);
#endif

  reconstruct_energy arch_args (N_s, N_E, slicenum, rhoeLim, KLim, EMono, ELim);
}

extern "C"
void Linternal_energy_multifluid(int N_s, int N_E, int slicenum, scalar* p, scalar* g, scalar* rhoe){
  /*!
    \brief Host C function to lauch internal_energy_multifluid kernel.
    \param[in] N_s number of nodes per element
    \param[in] N_E number of elements
    \param[in] slicenum to decompose higher dimensional problem into 1D slices
    \param[in] p monomial pressure solution
    \param[in] g monomial 1/(gamma-1) solution
    \param[out] rhoe monomial internal energy
    \section Description
    In GPU mode, launches N_E/blkE blocks of N_s x slicenum x blkE
    threads. blkE controls the number of elements to set on each block
  */
#ifdef USE_GPU
  int div = N_E/blkE;
  int mod = 0;
  if (N_E%blkE != 0) mod = 1;
  dim3 dimBlock(N_s,slicenum,blkE);
  dim3 dimGrid(div+mod,1);
#endif

  internal_energy_multifluid arch_args (N_s, N_E, slicenum, p, g, rhoe);

}

extern "C"
void Linternal_energy_stiffened(int N_s, int N_E, int slicenum, scalar* p, scalar* g, scalar* b, scalar* rhoe){
  /*!
    \brief Host C function to lauch internal_energy_multifluid kernel.
    \param[in] N_s number of nodes per element
    \param[in] N_E number of elements
    \param[in] slicenum to decompose higher dimensional problem into 1D slices
    \param[in] p monomial pressure solution
    \param[in] g monomial 1/(gamma-1) solution
    \param[in] b monomial gamma*pinf/(gamma-1) solution
    \param[out] rhoe monomial internal energy
    \section Description
    In GPU mode, launches N_E/blkE blocks of N_s x slicenum x blkE
    threads. blkE controls the number of elements to set on each block
  */
#ifdef USE_GPU
  int div = N_E/blkE;
  int mod = 0;
  if (N_E%blkE != 0) mod = 1;
  dim3 dimBlock(N_s,slicenum,blkE);
  dim3 dimGrid(div+mod,1);
#endif

  internal_energy_stiffened arch_args (N_s, N_E, slicenum, p, g, b, rhoe);
}

extern "C"
void Lhrl1D(int N_s, int N_E, int Nfields, int N_N, int slicenum, int* neighbors, int offxy, scalar* A, scalar* Alim){
  /*!
    \brief Host C function to launch hrl1D kernel.
    \param[in] N_s number of nodes per element
    \param[in] N_E number of elements
    \param[in] Nfields number of fields to operate on (eg. one field instead of N_F)
    \param[in] N_N number of neighbors per element
    \param[in] slicenum to decompose higher dimensional problem into 1D slices
    \param[in] neighbors array containing an element's neighbors
    \param[in] offxy offset if limiting in x or y
    \param[in] A solution to limit (monomial form)
    \param[out] Alim limited solution (monomial form)
    \section Description
    In GPU mode, launches N_E/blkE blocks of slicenum x Nfields x blkE
    threads. blkE controls the number of elements to set on each block
  */
#ifdef USE_GPU
  int div = N_E/blkE;
  int mod = 0;
  if (N_E%blkE != 0) mod = 1;
  dim3 dimBlock(slicenum,Nfields,blkE);
  dim3 dimGrid(div+mod,1);
#endif

  hrl1D arch_args (N_s, N_E, Nfields, N_N, slicenum, neighbors, offxy, A, Alim);
}

extern "C" 
  void Lhri1D(int N_s, int N_E, int N_N, int* neighbors, int N_s1D, int slicenum, int offxy, scalar* Lag2Mono, scalar* Mono2Lag, int* sensor, scalar* U, scalar* Unew){
  /*!
    \brief HR limiting function for individual elements (assumes 1D decomposition)
    \param[in] N_s number of nodes per element
    \param[in] N_E number of elements
    \param[in] N_N number of neighbors per element
    \param[in] neighbors array containing an element's neighbors
    \param[in] N_s1D number of nodes in a slice (1D element)
    \param[in] slicenum number of slices (in 2D N_s1D = slicenum)
    \param[in] offxy offset if limiting in x or y
    \param[in] sensors array of sensors
    \param[in] U solution to limit (Lagrange form)
    \param[out] Unew limited solution (only some may be limited bc of sensor)
    \section Description
    In GPU mode, launches N_E blocks of 1 x N_F x 1 threads. 
  */ 

#ifdef USE_GPU
  dim3 dimBlock(1,N_F,1);
  dim3 dimGrid(N_E,1);
#endif
  
  hri1D arch_args_array(N_F*(2*N_s*N_s + 7*N_s)*sizeof(scalar)) (N_s, N_E, N_N, neighbors, N_s1D, slicenum, offxy, Lag2Mono, Mono2Lag, sensor, U, Unew);
}

extern "C" 
void Lm2i1D(int N_s, int N_E, int N_N, int* neighbors, int N_s1D, int slicenum, int offxy, scalar* Lag2Mono, scalar* Mono2Lag, int* sensors, scalar* U, scalar* Unew){
  /*!
    \brief Modified limiting function for individual elements (assumes 1D decomposition)
    \param[in] N_s number of nodes per element
    \param[in] N_E number of elements
    \param[in] N_N number of neighbors per element
    \param[in] neighbors array containing an element's neighbors
    \param[in] N_s1D number of nodes in a slice (1D element)
    \param[in] slicenum number of slices (in 2D N_s1D = slicenum)
    \param[in] offxy offset if limiting in x or y
    \param[in] sensors array of sensors
    \param[in] U solution to limit (Lagrange form)
    \param[out] Unew limited solution (only some may be limited bc of sensor)
    \section Description
    In GPU mode, launches N_E blocks of 1 x 1 x 1 threads.
  */

#ifdef USE_GPU
  dim3 dimBlock(1,1,1);
  dim3 dimGrid(N_E,1);
#endif

  int size_share = 2*N_s*N_s + // for L2M/M2L
    N_F*3*N_s +  // room for L/C/R for all the fields
    N_s +  // for gammaLim
#ifdef STIFFENED
    N_s + // for betaLim
#endif 
    N_s +  // for pressureLim
    3*N_s + // for E L/C/R
    N_s + // for KLim
    N_s + // for rhoeLim
    4*N_s; // for tmp

  m2i1D arch_args_array(size_share*sizeof(scalar)) (N_s, N_E, N_N, neighbors, N_s1D, slicenum, offxy, Lag2Mono, Mono2Lag, sensors, U, Unew);
}

extern "C"
void Lhrl2D(int N_s, int N_E, int N_G, int N_N, int order, scalar* XYZCen, scalar* powersXYZG, int* neighbors, int* TaylorDxIdx, int* TaylorDyIdx, scalar* weight, scalar refArea, scalar* A, scalar* Alim){
  /*!
    \brief Host C function to launch hrl2D kernel.
    \param[in] N_s number of nodes per element
    \param[in] N_E number of elements
    \param[in] N_G number of gaussian nodes per element
    \param[in] N_N number of neighbors per element
    \param[in] order DG order
    \param[in] XYZCen element centroid coordinates
    \param[in] powersXYZG powers of coordinates of gaussian nodes
    \param[in] neighbors array containing an element's neighbors
    \param[in] TaylorDxIdx indices of Taylor polynomial derivatives in x
    \param[in] TaylorDyIdx indices of Taylor polynomial derivatives in y
    \param[in] weight gaussian integration weights
    \param[in] refArea area of reference triangle
    \param[in] A solution to limit (monomial form)
    \param[out] Alim limited solution (monomial form)
    \section Description
    In GPU mode, launches N_E/blkE blocks of 1 x N_F x blkE
    threads. blkE controls the number of elements to set on each block
  */
#ifdef USE_GPU
  int div = N_E/blkE;
  int mod = 0;
  if (N_E%blkE != 0) mod = 1;
  dim3 dimBlock(1,N_F,blkE);
  dim3 dimGrid(div+mod,1);
#endif
  
  hrl2D arch_args (N_s, N_E, N_G, N_N, order, XYZCen, powersXYZG, neighbors, TaylorDxIdx, TaylorDyIdx, weight, refArea, A, Alim);
}

extern "C"
void LChangeBasis(int size1, int size2, int N_E, scalar* Transform, scalar* U, scalar* Unew){
  /*!
    \brief Host C function to launch ChangeBasis kernel.
    \param[in] size1 number of rows of tranform
    \param[in] size2 number of columns of tranform
    \param[in] N_E number of elements
    \param[in] Transform Basis transform matrix (per element)
    \param[in] U solution to transform
    \param[out] Unew transformed solution
    \section Description
    In GPU mode, launches N_E/blkE blocks of size1 x N_F x blkE
    threads. blkE controls the number of elements to set on each block
  */
#ifdef USE_GPU
  int div = N_E/blkE;
  int mod = 0;
  if (N_E%blkE != 0) mod = 1;
  dim3 dimBlock(size1,N_F,blkE);
  dim3 dimGrid(div+mod,1);
#endif

  ChangeBasis arch_args (size1, size2, N_E, Transform, U, Unew);
}

//==========================================================================
//
//  Internal functions
//
//==========================================================================


 
//==========================================================================
//
//  Limiter functions
//
//==========================================================================



arch_device void getTaylorDerivative(int order, int N_s, scalar* T, int mx, int my, int* DxIdx, int* DyIdx, scalar* ddT){
  /*!
    \brief Not used right now
  */
//   // Get mx+my order derivative of Taylor polynomial T
//   int offsetx  = 0, offsety = 0;
//   scalar* dT = new scalar[N_s]; for(int i=0; i < N_s; i++) dT[i] = 0;

//   // Find the offsets
//   for(int p = order; p > order-mx; p--) offsetx += (p+1)*(p+2)/2;
//   for(int p = order; p > order-my; p--) offsety += (p+1)*(p+2)/2;
  
//   // Get mx-order order derivative wrt x
//   for(int k = 0; k < (order-mx+1)*(order-mx+2)/2; k++){
//     dT[k] = T[DxIdx[offsetx + k]];
//   }

//   // Get my-order order derivative wrt y
//   for(int k = 0; k < (order-my+1)*(order-my+2)/2; k++){
//     ddT[k] = dT[DyIdx[offsety + k]];
//   }

//   delete[] dT;
}

arch_device scalar CellAvg(int N_G, int ioff, scalar* weight, scalar refArea, scalar* powers, int N_s, scalar* T){
  /*!
    \brief Not used right now. Get cell avg of a polynomial of order=order in a cell
    \return cell average
    \section Description
    ioff = 0 for full polynomial
    ioff = 3 for remainder polynomial
  */  
  scalar I = 0;
  scalar w = 0;
  for(int g = 0; g < N_G; g++){
    w = weight[g];
    for(int i = ioff; i < N_s; i++){
      I += T[i]*w*powers[g*N_s+i];
    }
  }


  // We want the cell average:
  // avg = \frac{1}{|\Omega|} \int_\Omega U(x,y) \ud \Omega
  //     = \frac{1}{|\Omega|} \sum_k w_k U(x_k,y_k) J
  //     = \frac{1}{J |\omega|} \sum_k w_k U(x_k,y_k) J
  //     = \frac{1}{|\omega|} \sum_k w_k U(x_k,y_k)
  return I/refArea; // omega = 1/2 for a triangle
}


//==========================================================================
arch_device scalar minmod(scalar a, scalar b){
  /*!
    \brief Minmod function for 2 arguments
    \param[in] a first arg
    \param[in] b second arg
    \return minmod(a,b)
    \section Description
    eq 2.19 of "Hierarchical reconstruction for discontinuous Galerkin methods..."
  */
  int signa = signum(a);
  if (signa != signum(b)) return 0;

  scalar fabsa = fabs(a);
  scalar fabsb = fabs(b);
  if (fabsa<fabsb) return signa*fabsa;
  else return signa*fabsb;
}

//==========================================================================
arch_device inline scalar minabs(scalar* c, int n){
  /*!
    \brief Minimum of the absolute value of c
    \param[in] c array to find minabs of
    \param[in] n number of elements in c
    \return minabs of c
  */
  scalar minabs = fabs(c[0]);
  for(int i=1;i<n;i++) if (minabs>fabs(c[i])) minabs = fabs(c[i]);
  return minabs;
}

//==========================================================================
arch_device scalar minmod(scalar* c, int n){
  /*!
    \brief Generalized minmod function
    \param[in] c array to find minmod of
    \param[in] n number of elements in c
    \return minmod of c
    \section Description
    eq 2.19 of "Hierarchical reconstruction for discontinuous Galerkin methods..."
  */
  int sign = signum(c[0]);
  for(int i=1; i<n; i++){
    if (sign!=signum(c[i])) return 0;
  }
  return sign*minabs(c,n);
}

//==========================================================================
arch_device scalar minmod2(scalar* c, int n){
  /*!
    \brief Generalized minmod function (alternate)
    \param[in] c array to find minmod of
    \param[in] n number of elements in c
    \return minmod of c
    \section Description
    eq 2.20 of "Hierarchical reconstruction for discontinuous Galerkin methods..."
  */
  scalar min = c[0];
  for(int i=1; i<n; i++) if(fabs(c[i])<fabs(min)) min = c[i];
  return min;
}

//==========================================================================
arch_device void limit_monomial(int N, scalar* AL, scalar* AC, scalar* AR, scalar* Alim){
  /*!
    \brief Limit a 1D monomial using HR
    \param[in] N monomial order
    \param[in] AL left cell monomial
    \param[in] AC center cell monomial
    \param[in] AR right cell monomial
    \param[out] Alim limited center cell monomial
  */

  scalar avgdUL = 0, avgdUC=0, avgdUR=0; scalar integral = 0;
  scalar avgRL = 0, avgRC=0, avgRR=0; scalar alim = 0;
  scalar avgLL = 0, avgLC=0, avgLR=0;
  scalar c1,c2;

  // Loop on derivatives
  for(int m = N; m > 0; m--){
    avgdUL = 0; avgdUC=0; avgdUR=0;
    avgRL = 0; avgRC = 0; avgRR = 0;

    // Calculate the derivative average in the cells: left,
    // center, right. Calculate the remainder polynomial in our
    // cells and its two neighbors
    for(int n=m-1; n<=N; n++){
      integral = integrate_monomial_derivative(m-1,n);
      avgdUL += AL[n]*integral;
      avgdUC += AC[n]*integral;
      avgdUR += AR[n]*integral;
      if(n>=m+1){
	alim = Alim[n];
	avgRL += alim*integrate_monomial_derivative_bounds(m-1,n,-3,-1);
	avgRC += alim*integral;
	avgRR += alim*integrate_monomial_derivative_bounds(m-1,n,1,3);
      }
    }
	  
    // Approximate the average of the linear part
    avgLL = 0.5*(avgdUL - avgRL); // avg = \frac{1}{2} \int_{-1}^1 U \ud x
    avgLC = 0.5*(avgdUC - avgRC);
    avgLR = 0.5*(avgdUR - avgRR);
	
    // MUSCL approach to get candidate coefficients
    c1 = 0.5*(avgLC - avgLL);  // 1/dx = 1/2 = 0.5
    c2 = 0.5*(avgLR - avgLC);

    // Limited value
    Alim[m] = minmod(c1,c2); 
  }// end loop on m
  Alim[0] = avgLC;
}


//==========================================================================
arch_device int lim_factorial(int n)
{
  /*!
    \brief Factorial function
    \param[in] n get factorial of this number
    \return factorial of n
  */
  if     (n== 0) return 1;
  else if(n== 1) return 1;
  else if(n== 2) return 2;
  else if(n== 3) return 6;
  else if(n== 4) return 24;
  else if(n== 5) return 120;
  else if(n== 6) return 720;
  else if(n== 7) return 5040;
  else if(n== 8) return 40320;
  else if(n== 9) return 362880;
  else if(n==10) return 3628800;
  else if(n==11) return 39916800;
  else if(n==12) return 479001600;
  return 1; // default return for lim_factorial
}

//==========================================================================
arch_device int binomial_coefficient(int n, int k){
  /*!
    \brief Binomial coefficient function
    \param[in] n
    \param[in] k
    \return C(n,k)
    \section Description
    Inspired from https://gist.github.com/jeetsukumaran/5392166.
    Does not handle super large numbers (no need really)
  */

  if (0 == k || n == k) {
    return 1;
  }
  if (k > n) {
    return 0;
  }
  if (k > (n - k)) {
    k = n - k;
  }
  if (1 == k) {
    return n;
  }
  int b = 1;
  for (int i = 1; i <= k; ++i) {
    b *= (n - (k - i));
    if (b < 0) return -1; /* Overflow */
    b /= i;
  }
  return b;
}

//==========================================================================
arch_device void gemm(int M, int N, int K, scalar* A, scalar* B, scalar*C){
  /*!
    \brief Matrix-matrix mutliplication C = A*B
    \param[in] M rows of A = rows of C
    \param[in] N columns of B = columns of C
    \param[in] K columns of A = rows of B
    \param[in] A first matrix
    \param[in] B second matrix
    \param[out] C C=A*B
    Assume column major order. Modeled on BLAS gemm.
  */
  scalar sum = 0;
  for(int m=0; m<M; m++){
    for(int n=0; n<N; n++){
      for(int k=0; k<K; k++){
	sum += A[k*M+m]*B[n*K+k];
      }
      C[n*M+m] = sum; sum = 0;
    }
  }
}

//==========================================================================
arch_device void gemm3(int M, int N, int K, scalar* A, scalar* B1, scalar*C1, scalar* B2, scalar*C2, scalar* B3, scalar*C3){
  /*!
    \brief Three matrix-matrix mutliplications: C1 = A*B1, C2 = A*B2, C3 = A*B3
    \param[in] M rows of A = rows of C
    \param[in] N columns of B = columns of C
    \param[in] K columns of A = rows of B
    \param[in] A first matrix
    \param[in] B1 second matrix (first gemm)
    \param[in] B2 second matrix (second gemm)
    \param[in] B3 second matrix (third gemm)
    \param[out] C1 C=A*B1
    \param[out] C2 C=A*B2
    \param[out] C3 C=A*B3
    Assume column major order. Modeled on BLAS gemm.
  */
  scalar sum1=0, sum2=0, sum3=0;
  scalar a;
  for(int m=0; m<M; m++){
    for(int n=0; n<N; n++){
      for(int k=0; k<K; k++){
	a = A[k*M+m]; // avoid mem fetches for all 3 products
	sum1 += a*B1[n*K+k];
	sum2 += a*B2[n*K+k];
	sum3 += a*B3[n*K+k];
      }
      C1[n*M+m] = sum1; sum1 = 0;
      C2[n*M+m] = sum2; sum2 = 0;
      C3[n*M+m] = sum3; sum3 = 0;
    }
  }
}

arch_device void gemv(int M, int N, scalar* A, scalar* B, scalar*C){
  /*!
    \brief Matrix-vector mutliplication C = A*B
    \param[in] M rows of A = rows of C
    \param[in] N columns of A = rows of B
    \param[in] A first matrix
    \param[in] B second matrix
    \param[out] C C=A*B
    Assume column major order. Modeled on BLAS gemv.
  */
  scalar sum = 0;
  for(int m=0; m<M; m++){
    for(int n=0; n<N; n++){
      sum += A[n*M+m]*B[n];
    }
    C[m] = sum; sum = 0;
  }
}

//==========================================================================
arch_device void gemv3(int M, int N, scalar* A, scalar* B1, scalar*C1, scalar* B2, scalar*C2, scalar* B3, scalar*C3){
  /*!
    \brief Three matrix-vector mutliplications: C1 = A*B1, C2 = A*B2, C3 = A*B3
    \param[in] M rows of A = rows of C
    \param[in] N columns of A = rows of B
    \param[in] A first matrix
    \param[in] B1 second matrix (first gemv)
    \param[in] B2 second matrix (second gemv)
    \param[in] B3 second matrix (third gemv)
    \param[out] C1 C=A*B1
    \param[out] C2 C=A*B2
    \param[out] C3 C=A*B3
    Assume column major order. Modeled on BLAS gemv.
  */
  scalar sum1=0, sum2=0, sum3=0;
  scalar a;
  for(int m=0; m<M; m++){
    for(int n=0; n<N; n++){
      a = A[n*M+m]; // avoid mem fetches for all 3 products
      sum1 += a*B1[n];
      sum2 += a*B2[n];
      sum3 += a*B3[n];
    }
    C1[m] = sum1; sum1 = 0;
    C2[m] = sum2; sum2 = 0;
    C3[m] = sum3; sum3 = 0;
  }
}

arch_device inline scalar integrate_monomial_derivative(int k, int n)
{
  /*!
    \brief The integral of the kth derivative of nth order monomial (from -1 to 1)
    \param[in] k kth derivative of the polynomial
    \param[in] n monomial order
    \return \frac{2}{(n-k+1)!} if n-k+1 is odd, 0 otherwise
    Calculates $\int_{-1}^1 \frac{\partial^k}{\partialx^k} \frac{x^n}{n!} \mathrm{d} x$
  */
  int num = n-k+1;
  if (num%2) return 2.0/(scalar)lim_factorial(num);
  else return 0.0;
}

arch_device inline scalar integrate_monomial_derivative_bounds(int k, int n, scalar a, scalar b)
{
  /*!
    \brief The integral of the kth derivative of nth order monomial.
    \param[in] k kth derivative of the polynomial
    \param[in] n monomial order
    \param[in] a lower integral bound
    \param[in] b upper integral bound
    \return the integral
    Calculates $\int_{a}^{b} \frac{\partial^k}{\partialx^k} \frac{x^n}{n!} \mathrm{d} x$
  */
  int num = n-k+1;
  return (pow(b,num) - pow(a,num))/(scalar)lim_factorial(num);
}

arch_device void average_monomial(int N, scalar* A, scalar* Alim){
  /*!
    \brief Given a monomial, make all the slopes 0, return the average
    \param[in] N monomial order
    \param[in] A monomial (1D)
    \param[out] Alim monomial set to average in cell (slopes=0)
  */
  scalar avg = 0;
  for(int n=0; n<=N; n++){
    avg += A[n]*integrate_monomial_derivative(0,n);
    Alim[n] = 0;  // set slopes to 0
  }
  Alim[0] = 0.5*avg;
}

arch_device void set2average(int N_s, int N, int N_s1D, int slicenum, scalar* L2M, scalar* M2L, scalar* tmp, scalar* U, scalar* UMonoLim){
  /*!
    \brief Set a nodal solution U to its cell average
    \param[in] N_s number of nodes per element
    \param[in] N 1D monomial order
    \param[in] N_s1D number of nodes in 1D elemement
    \param[in] slicenum number of slices
    \param[in] L2M Lagrange -> Monomial transform
    \param[in] M2L Monomial -> Lagrange transform
    \param[in] tmp temporary array to store intermediate values
    \param[out] U solution to be averaged
    \param[out] UMonoLim (optional) array to hold limited monomial values
  */

  // Initializations
  int cnt = 0;
  scalar* A    = &tmp[cnt]; cnt+=N_s;
  scalar* Alim = &tmp[cnt]; 

  // Lagrange -> monomial transform
  gemv(N_s,N_s,L2M,U,A);

  // Limit
  for(int slice=0; slice<slicenum; slice++){
    int idx = slice*N_s1D;
    average_monomial(N,&A[idx],&Alim[idx]);
  }

  // Monomial -> lagrange transform
  gemv(N_s,N_s,M2L,Alim,U);

  // If desired, also return the limited monomial values
  if(UMonoLim != NULL){for(int i=0; i<N_s; i++){UMonoLim[i] = Alim[i];}}
  
  A = NULL;  Alim = NULL;  
}

arch_device void HR(int N_s, int N, int N_s1D, int slicenum, scalar* L2M, scalar* M2L, scalar* tmp, scalar* UL, scalar* UC, scalar* UR, scalar* UMonoLim){
  /*!
    \brief Limit a nodal solution U using HR (1D decomposition)
    \param[in] N_s number of nodes per element
    \param[in] N 1D monomial order
    \param[in] N_s1D number of nodes in 1D elemement
    \param[in] slicenum number of slices
    \param[in] L2M Lagrange -> Monomial transform
    \param[in] M2L Monomial -> Lagrange transform
    \param[in] tmp temporary array to store intermediate values
    \param[in] UL solution on the left
    \param[out] UC solution to be averaged
    \param[in] UR solution on the right
    \param[out] UMonoLim (optional) array to hold limited monomial values
  */

  // Initializations
  int cnt = 0;
  scalar* AL   = &tmp[cnt]; cnt+=N_s;
  scalar* AC   = &tmp[cnt]; cnt+=N_s;
  scalar* AR   = &tmp[cnt]; cnt+=N_s;
  scalar* Alim = &tmp[cnt]; 

  // Lagrange -> monomial transform
  gemv3(N_s,N_s,L2M,UL,AL,UC,AC,UR,AR);

  // Limit
  for(int slice=0; slice<slicenum; slice++){
    int idx = slice*N_s1D;
    limit_monomial(N,&AL[idx],&AC[idx],&AR[idx],&Alim[idx]);
  }

  // Monomial -> lagrange transform
  gemv(N_s,N_s,M2L,Alim,UC);
  
  // If desired, also return the limited monomial values
  if(UMonoLim != NULL){for(int i=0; i<N_s; i++){UMonoLim[i] = Alim[i];}}

  AR = NULL; AC = NULL; AL = NULL;  Alim = NULL;
}

arch_device void pressure(int N_s, scalar* rho, scalar* rhou, scalar* rhov, scalar* E, scalar* gamma, scalar* beta, scalar* p){
  /*!
    \brief Get the pressure in an individual element
    \param[in] N_s number of nodes in an element
    \param[in] rho density
    \param[in] rhou x-momentum
    \param[in] rhov y-momentum
    \param[in] E total energy
    \param[in] gamma 1/(gamma-1)
    \param[in] beta pinf*gamma/(gamma-1)
    \param[out] p pressure
  */
  for(int i=0; i<N_s; i++){
#ifdef ONED
#ifdef MULTIFLUID
    p[i] = (E[i] - 0.5*rhou[i]*rhou[i]/rho[i])/gamma[i];
#elif STIFFENED
    p[i] = (E[i] - beta[i] - 0.5*rhou[i]*rhou[i]/rho[i])/gamma[i];
#endif 
#elif TWOD
#ifdef MULTIFLUID
    p[i] = (E[i] - 0.5*(rhou[i]*rhou[i]+rhov[i]*rhov[i])/rho[i])/gamma[i];
#elif STIFFENED
    p[i] = (E[i] - beta[i] - 0.5*(rhou[i]*rhou[i]+rhov[i]*rhov[i])/rho[i])/gamma[i];
#endif
#endif    
  }
}

arch_device void kinetic_energy(int N_s, scalar* L2M, scalar* rho, scalar* rhou, scalar* rhov, scalar* tmp, scalar* K){
  /*!
    \brief Get the kinetic energy in monomial form
    \param[in] N_s number of nodes per element
    \param[in] L2M Lagrange -> Monomial transform
    \param[in] rho density (Lagrange form)
    \param[in] rhou x-momentum (Lagrange form)
    \param[in] rhov y-momentum (Lagrange form)
    \param[in] tmp temporary array to store intermediate values
    \param[out] K kinetic energy (Monomial form)
  */

  // Nodal kinetic energy stored in tmp
  for(int i=0; i<N_s; i++){
#ifdef ONED
    tmp[i] = 0.5*(rhou[i]*rhou[i])/rho[i];
#elif TWOD
    tmp[i] = 0.5*(rhou[i]*rhou[i]+rhov[i]*rhov[i])/rho[i];
#endif
  }

  // Lagrange -> monomial transform
  gemv(N_s,N_s,L2M,tmp,K);
}

arch_device void internal_energy(int N_s1D, int slicenum, scalar* p, scalar* g, scalar* b, scalar* rhoe){
  /*!
    \brief Reconstruct the internal energy in monomial form
    \param[in] N_s1D number of nodes in 1D elemement
    \param[in] slicenum number of slices
    \param[in] p monomial pressure solution
    \param[in] g monomial 1/(gamma-1) solution
    \param[in] b monomial gamma*pinf/(gamma-1) solution
    \param[out] rhoe monomial internal energy
    \section Description
    Reconstruct the monomial internal energy coefficients using the
    pressure, 1/gamma-1, and gamma*pinf/(gamma-1) coefficients so that
    the pressure remains non-oscillatory
  */

  for(int slice=0; slice<slicenum; slice++){
    for(int i=0; i<N_s1D; i++){

      scalar I = 0;
      for(int k=0; k<i+1; k++){
	I += (scalar)binomial_coefficient(i,k) * p[slice*N_s1D+i-k] * g[slice*N_s1D+k];
      }
#ifdef STIFFENED
      I += b[slice*N_s1D+i];
#endif
      rhoe[slice*N_s1D+i] = I;
    } // loop on N_s1D
  } // loop on slices
}


arch_device void reconstruct_total_energy(int N_s, int N_s1D, int slicenum, scalar* L2M, scalar* M2L, scalar* rhoeLim, scalar* KLim, scalar* tmp, scalar* E){
    /*!
    \brief Reconstruct the energy lagrange polynomial for an individual element
    \param[in] N_s number of nodes per element
    \param[in] N_s1D number of nodes per slice
    \param[in] rhoeLim limited monomial internal energy
    \param[in] KLim limited monomial kinetic energy
    \param[out] E limited total energy (lagrange form)
    \section Description
    Reconstruct the energy Lagrange coefficients using the internal
    and kinetic energy monomial coefficients. Apply a correction to the
    zeroth coefficients so that the method is conservative.
  */

  int cnt = 0;
  scalar* EMono = &tmp[cnt]; cnt+=N_s;
  scalar* ELim  = &tmp[cnt];

  // Lagrange -> monomial transform
  gemv(N_s,N_s,L2M,E,EMono);

  // Reconstruct the total energy   
  int idx= 0, idx0 = 0;
  for(int slice = 0; slice < slicenum; slice++){
    idx0 = slice*N_s1D + 0;

    // Start at idx 1 because we will do the zeroth coefficient separately
    for(int i = 1; i < N_s1D; i++){
      idx = slice*N_s1D+i;
      ELim[idx] = rhoeLim[idx]+KLim[idx];
    }

    // Correct the zeroth coefficient to conserve energy
    scalar E0 = EMono[idx0];
    for(int k = 2; k<N_s1D; k+=2){
      idx = slice*N_s1D+k;
      E0 -= 1.0/((scalar)lim_factorial(k+1)) * (ELim[idx]-EMono[idx]);
    }
    ELim[idx0] = E0;

  } // loop on slices

  // Monomial -> lagrange transform
  gemv(N_s,N_s,M2L,ELim,E);

  EMono = NULL; ELim = NULL;
}
