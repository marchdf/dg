
#include <hip/hip_runtime.h>
/*!
  \file printer_oned_stiffened.cu
  \brief Kernel to output 1D stiffened solution used by the PRINTER class
  \copyright Copyright (C) 2014, Regents of the University of Michigan
  \author Marc T. Henry de Frahan <marchdf@umich.edu>, Computational Flow Physics Laboratory, University of Michigan
  \ingroup printer
*/
#ifdef ONED
#ifdef STIFFENED
#include <printer_kernels.h>

//==========================================================================
//
// Kernel definitions
//
//==========================================================================

//==========================================================================
arch_global void formater(int N_s, int N_E, scalar* U, scalar* output, bool inverse){
  /*!
    \brief Format solution kernel.
    \param[in] N_s number of nodes per element
    \param[in] N_E number of elements
    \param[in] U solution to format to output
    \param[out] output output solution array
    \param[in] inverse true if you want to copy from output to U instead
  */

  //
  // Copy from U to output
  //
  if(!inverse){
#ifdef USE_CPU
    for(int e = 0; e < N_E; e++){
      for(int i = 0; i < N_s; i++){
#elif USE_GPU
    int e = blockIdx.x*blkE+threadIdx.z;
    if (e < N_E){
      int i = threadIdx.x;{
#endif

	// Separate the fields
	scalar rho = U[(e*N_F+0)*N_s+i];
	scalar ux  = U[(e*N_F+1)*N_s+i]/rho;
	scalar et  = U[(e*N_F+2)*N_s+i];
	scalar gamma = 1+1.0/U[(e*N_F+3)*N_s+i];
	scalar pinf = (1-1.0/gamma)*U[(e*N_F+4)*N_s+i];
	
	output[(e*N_F+0)*N_s+i] = rho;
	output[(e*N_F+1)*N_s+i] = ux;
	output[(e*N_F+2)*N_s+i] = gamma;
	output[(e*N_F+3)*N_s+i] = pinf;
	output[(e*N_F+4)*N_s+i] = (gamma-1)*(et - gamma*pinf/(gamma-1) - 0.5*ux*ux*rho);
	
	// Mass fractions
#include "loopstart.h"
#define LOOP_END N_Y
#define MACRO(x) output[(e*N_F+5+x)*N_s+i] = U[(e*N_F+5+x)*N_s+i]/rho;
#include "loop.h"
      } // loop on nodes
    } // loop on elements
  } // if inverse

  //
  // Copy from output to U
  //
  else {
#ifdef USE_CPU
    for(int e = 0; e < N_E; e++){
      for(int i = 0; i < N_s; i++){
#elif USE_GPU
    int e = blockIdx.x*blkE+threadIdx.z;
    if (e < N_E){
      int i = threadIdx.x;{
#endif

	// Get fields from output
	scalar rho   = output[(e*N_F+0)*N_s+i];
	scalar ux    = output[(e*N_F+1)*N_s+i];
	scalar gamma = output[(e*N_F+2)*N_s+i];
	scalar pinf  = output[(e*N_F+3)*N_s+i];
	scalar p     = output[(e*N_F+4)*N_s+i];

	U[(e*N_F+0)*N_s+i] = rho;
	U[(e*N_F+1)*N_s+i] = rho*ux;
	U[(e*N_F+2)*N_s+i] = p/(gamma-1) + gamma*pinf/(gamma-1) + 0.5*rho*ux*ux;
	U[(e*N_F+3)*N_s+i] = 1.0/(gamma-1);
	U[(e*N_F+4)*N_s+i] = gamma*pinf/(gamma-1);

	// Mass fractions
#include "loopstart.h"
#define LOOP_END N_Y
#define MACRO(x) U[(e*N_F+5+x)*N_s+i] = rho*output[(e*N_F+5+x)*N_s+i];
#include "loop.h"
      } // loop on nodes
    }  // loop on elements
  } // if inverse
}


//==========================================================================
//
//  Host C functions
//
//==========================================================================

extern "C"
void Lformater(int N_s, int N_E, scalar* U, scalar* output, bool inverse){
  /*!
    \brief Host C function to lauch format kernel.
    \param[in] N_s number of nodes per element
    \param[in] N_E number of elements
    \param[in] U solution to format to output
    \param[out] output output solution array
    \param[in] inverse true if you want to copy from output to U instead (default false)
    \section Description
    In GPU mode, launches N_E/blkE blocks of N_s x 1 x blkE
    threads. blkE controls the number of elements to set on each block
  */
#ifdef USE_GPU
  int div = N_E/blkE;
  int mod = 0;
  if (N_E%blkE != 0) mod = 1;
  dim3 dimBlock(N_s,1,blkE);
  dim3 dimGrid(div+mod,1);
#endif

  formater arch_args (N_s, N_E, U, output, inverse);
};
#endif
#endif
